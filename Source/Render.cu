#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <glm/glm.hpp>
#include <glm/gtx/compatibility.hpp>
#include <algorithm>
#include <thrust/device_new.h>
#include <thrust/device_delete.h>
#include <iostream>

#include "Typedef.cuh"
#include "Camera.cuh"
#include "Hit/HittableObjects.cuh"
#include "Hit/HittableObjectsDevice.cuh"
#include "Util/Random.cuh"
#include "Surface.cuh"
#include "Render.cuh"

__global__ void CUDA_render_init(CUDARenderer renderer);

__global__ void CUDA_render_render(CUDARenderer renderer);

__device__ Color CUDA_rayColor(const Ray& ray,
    const HittableObjectsDevice& world,
    hiprandState* localRandomState,
    int framebufferIdx,
    int maxRecursionDepth)
{
    float currentAttenuation = 1.0f;
    Ray currentRay = ray;
    for (int i = 0; i < maxRecursionDepth; ++i)
    {
        HitData hitData;
        bool hasValue = false;
        world.hit(hitData, hasValue, currentRay, 0.001f, std::numeric_limits<float>::infinity());
        if (hasValue)
        {
            auto target = hitData.coord + hitData.N
                + glm::normalize(randomInHemisphere(hitData.N, localRandomState, framebufferIdx));
            currentAttenuation *= 0.5f;
            currentRay = Ray(hitData.coord, target - hitData.coord);
        }
        else
        {
            auto directionNorm = glm::normalize(currentRay.direction());
            auto t = (directionNorm.y + 1.0f) * 0.5f;
            auto color = glm::lerp(Color(0.67f, 0.84f, 0.92f), Color(1.0f), glm::vec3(t));
            return currentAttenuation * color;
        }
    }

    return Color(0.0f, 0.0f, 0.0f);
}

__global__ void CUDA_render_init(CUDARenderer renderer)
{
    auto framebufferIdx = renderer.framebufferIndex(blockIdx.x, threadIdx.x);

    hiprand_init(0, framebufferIdx, 0, &thrust::raw_pointer_cast(renderer.getRandomStates())[framebufferIdx]);
}

__global__ void CUDA_render_init_(hiprandState* randomState, int windowWidth, int windowHeight)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    int framebufferIdx = windowWidth * j + i;

    hiprand_init(0, framebufferIdx, 0, &randomState[framebufferIdx]);
}

__global__ void CUDA_render_render(CUDARenderer renderer)
{
    // DO NOT modify threadIdx.x or blockIdx.x by adding them directly, copy their value first!
    auto i = blockIdx.x;
    auto j = threadIdx.x;
    auto framebufferIdx = renderer.framebufferIndexFlipped(i, j);
    auto localRandomState = thrust::raw_pointer_cast(renderer.getRandomStates())[framebufferIdx];
    Color accColor(0.0f);

    for (int s = 0; s < renderer.getAntiAliasingPixelSamples(); s++)
    {
        float u = ((float)i + hiprand_uniform(&localRandomState)) / (float)renderer.getWidth();
        float v = ((float)j + hiprand_uniform(&localRandomState)) / (float)renderer.getHeight();
        accColor += CUDA_rayColor(renderer.getCamera().getRay(u, v), renderer.getWorld(), &localRandomState, framebufferIdx, renderer.getMaxLightBounce());
    }
    float colorScale = 1.0f / (float)renderer.getAntiAliasingPixelSamples();
    float gammaPower = 1.0f / renderer.getGamma();
    glm::vec3 colorScaled = glm::pow(accColor * colorScale, glm::vec3(gammaPower));

    int col = 0x00000000;
    col |= std::clamp(static_cast<int>(255.0f * colorScaled.r), 0, 255) << 16;
    col |= std::clamp(static_cast<int>(255.0f * colorScaled.g), 0, 255) << 8;
    col |= std::clamp(static_cast<int>(255.0f * colorScaled.b), 0, 255);

    thrust::raw_pointer_cast(renderer.getGPUFramebuffer())[framebufferIdx] = col;
}

__global__ void CUDA_render_render_(uint32_t* framebuffer,
    hiprandState* randomState,
    int windowWidth,
    int windowHeight,
    int pixelSamples,
    Camera camera,
    const HittableObjectsDevice world,
    int maxRecursionDepth)
{
    // DO NOT modify threadIdx.x or blockIdx.x by adding them directly, copy their value first!
    int i = blockIdx.x;
    int j = threadIdx.x;

    int framebufferIdx = windowWidth * (windowHeight - 1 - j) + i;

    auto localRandomState = randomState[framebufferIdx];

    Color accColor(0.0f);

    for (int s = 0; s < pixelSamples; s++)
    {
        float u = ((float)i + hiprand_uniform(&localRandomState)) / (float)windowWidth;
        float v = ((float)j + hiprand_uniform(&localRandomState)) / (float)windowHeight;
        accColor += CUDA_rayColor(camera.getRay(u, v), world, &localRandomState, framebufferIdx, maxRecursionDepth);
    }
    float gamma = 2.0f;
    float colorScale = 1.0f / (float)pixelSamples;
    float gammaPower = 1.0f / gamma;
    glm::vec3 colorScaled = glm::pow(accColor * colorScale, glm::vec3(gammaPower));

    int col = 0x00000000;
    col |= std::clamp(static_cast<int>(255.0f * colorScaled.r), 0, 255) << 16;
    col |= std::clamp(static_cast<int>(255.0f * colorScaled.g), 0, 255) << 8;
    col |= std::clamp(static_cast<int>(255.0f * colorScaled.b), 0, 255);

    framebuffer[framebufferIdx] = col;
}
