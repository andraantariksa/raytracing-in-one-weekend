#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <glm/glm.hpp>
#include <glm/gtx/compatibility.hpp>
#include <algorithm>
#include <iostream>

#include "typedef.cuh"
#include "Camera.cuh"
#include "Hit/HittableObjects.cuh"
#include "Hit/HittableObjectsDevice.cuh"
//#include "Misc/Random.hpp"
#include "Surface.cuh"
#include "Render.cuh"

CUDA_Render CUDA_render_setup(int width, int height, int pixelSamples)
{
    CUDA_Render cudaRender{};
    hipMalloc((void**)&cudaRender.gpuFramebuffer, sizeof(uint32_t) * width * height);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    hipMalloc((void**)&cudaRender.randomState, width * height * pixelSamples *sizeof(hiprandState));
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return cudaRender;
}

void CUDA_render_destroy(CUDA_Render* cudaRender)
{
    hipFree(cudaRender->gpuFramebuffer);
    hipFree(cudaRender->randomState);
}

__device__ Color CUDA_rayColor(const Ray& ray, const HittableObjectsDevice& world)
{
    HitData hitData;
    bool hasValue = false;
    world.hit(hitData, hasValue, ray, 0.0f, std::numeric_limits<float>::infinity());
    if (hasValue)
    {
        return 0.5f * Color(std::max(hitData.N.x + 1.0f, 1.0f),
            std::max(hitData.N.y + 1.0f, 1.0f),
            std::max(hitData.N.z + 1.0f, 1.0f));
    }

    auto directionNorm = glm::normalize(ray.direction());
    auto t = (directionNorm.y + 1.0f) * 0.5f;
    return glm::lerp(Color(0.67f, 0.84f, 0.92f), Color(1.0f), glm::vec3(t));
}

__global__ void CUDA_render_init(hiprandState* randomState, unsigned short section, int windowWidth, int windowHeight)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    int framebufferIdx = windowWidth * j + i;

    hiprand_init(0, framebufferIdx, 0, &randomState[framebufferIdx]);
}

__global__ void CUDA_render_render_(uint32_t* framebuffer, hiprandState* randomState, unsigned short section, int windowWidth, int windowHeight, int pixelSamples, Camera camera, const HittableObjectsDevice world, Surface& s)
{
    // DO NOT modify threadIdx.x or blockIdx.x by adding them directly, copy their value first!
    int i = blockIdx.x;
    int j = threadIdx.x;

    int framebufferIdx = windowWidth * j + i;

    Color accColor(0.0f);

    auto localRandomState = randomState[framebufferIdx];

    for (int s = 0; s < pixelSamples; s++)
    {
        int a = (int)std::ceilf(hiprand_uniform(&localRandomState) * 3.0f) - 2;
        float u = (float)(i + a) / (float)windowWidth;
        int b = (int)std::ceilf(hiprand_uniform(&localRandomState) * 3.0f) - 2;
        float v = (float)(j + b) / (float)windowHeight;
        accColor += CUDA_rayColor(camera.getRay(u, v), world);
    }
    Color colorScaled = accColor / (float)pixelSamples;

    int col = 0x00000000;
    col |= std::clamp(static_cast<int>(255.0f * colorScaled.r), 0, 255) << 16;
    col |= std::clamp(static_cast<int>(255.0f * colorScaled.g), 0, 255) << 8;
    col |= std::clamp(static_cast<int>(255.0f * colorScaled.b), 0, 255);

    framebuffer[framebufferIdx] = col;
}
