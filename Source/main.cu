﻿#include <SDL2/SDL.h>
#include <cassert>
#include <cmath>
#include <glm/glm.hpp>
#include <glm/gtx/transform.hpp>
#include <glm/gtx/string_cast.hpp>
#include <glm/gtx/compatibility.hpp>
#include <thrust/device_malloc.h>
#include <glm/common.hpp>
#include <utility>
#include <iostream>
#include <ctime>
#include <sstream>
#include <ppl.h>

#include "Typedef.cuh"
#include "Surface.cuh"
#include "Ray.cuh"
#include "Hit/HittableObjects.cuh"
#include "Hit/Object/SphereObject.cuh"
#include "Camera.cuh"
#include "Render.cuh"
#include "Util/Random.cuh"
#include "Hit/HittableObjectsDevice.cuh"
#include "Util/SDLHelpers.cuh"
#include "Util/Timer.cuh"

int main()
{
    assert(SDL_Init(SDL_INIT_VIDEO) == 0);

    const int windowWidth = 1280;
    const int windowHeight = 720;
    const float viewportWidth = 10.0f * ((float)windowWidth / (float)windowHeight);
    const float viewportHeight = 10.0f;
    const float vocalLength = 10.0f;
    int pixelSamples = 100;
    int maxRecursionDepth = 50;

    auto* window =
        SDL_CreateWindow("Raytracing", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, windowWidth, windowHeight, 0);
    assert(window != nullptr);

//    CUDA_Render cudaRender{};
//    hipMalloc((void**)&cudaRender.gpuFramebuffer, sizeof(uint32_t) * windowWidth * windowHeight);
//    printf("%s\n", hipGetErrorString(hipGetLastError()));
//    hipMalloc((void**)&cudaRender.randomState, sizeof(hiprandState) * windowWidth * windowHeight);
//    printf("%s\n", hipGetErrorString(hipGetLastError()));

    // Draw
    Surface surf(window, windowWidth, windowHeight);
    Camera camera(glm::vec3(0.0f), viewportWidth, viewportHeight, vocalLength);

//    HittableObjects world;
//    world.add(std::make_shared<SphereObject>(SphereObject(glm::vec3(0.0f, 0.0f, -10.0f), 3.0f)));
//    world.add(std::make_shared<SphereObject>(SphereObject(glm::vec3(-4.0f, 0.0f, -10.0f), 4.0f)));

    HittableObjectsDevice worldDevice(2);

    SphereObject* od_1;
    hipMalloc(&od_1, sizeof(SphereObject));
    SphereObject oh_1(glm::vec3(0.0f, 0.0f, -10.0f), 3.0f);
    hipMemcpy(od_1, &oh_1, sizeof(SphereObject), hipMemcpyHostToDevice);
    hipMemcpy(&worldDevice.m_objects[0], &od_1, sizeof(IHittableObject*), hipMemcpyHostToDevice);
//    worldDevice.set(0, (IHittableObject **)&od_1);

    SphereObject* od_2;
    hipMalloc(&od_2, sizeof(SphereObject));
    SphereObject oh_2(glm::vec3(-4.0f, 0.0f, -10.0f), 4.0f);
    hipMemcpy(od_2, &oh_2, sizeof(SphereObject), hipMemcpyHostToDevice);
    hipMemcpy(&worldDevice.m_objects[1], &od_2, sizeof(IHittableObject*), hipMemcpyHostToDevice);
//    worldDevice.set(1, (IHittableObject **)&od_2);

    CUDARenderer renderer(windowWidth, windowHeight, camera, worldDevice, pixelSamples, maxRecursionDepth, 2.0f);

//    CUDA_render_init_<<<1280, 720>>>(
//        cudaRender.randomState,
//        windowWidth,
//        windowHeight);
//    printf("%s\n", hipGetErrorString(hipGetLastError()));

    surf.setDrawFunc([&](auto s)
    {
//      CUDA_render_render_<<<1280, 720>>>(cudaRender.gpuFramebuffer,
//          cudaRender.randomState,
//          windowWidth,
//          windowHeight,
//          pixelSamples,
//          camera,
//          worldDevice,
//          maxRecursionDepth);
//        surf.copyFramebufferDeviceToHost(cudaRender.gpuFramebuffer);
        renderer.render();
        surf.copyFramebufferDeviceToHost(thrust::raw_pointer_cast(renderer.getGPUFramebuffer()));
    });
    surf.draw();

    // End draw

    Timer update;
    update.start();
    Timer fps;
    fps.start();

    SDL_Event event;
    int frame = 0;

    bool running = true;
    bool isNeedToRedraw = false;
    while (running)
    {
        while (SDL_PollEvent(&event))
        {
            switch (event.type)
            {
            case SDL_QUIT:
                running = false;
                break;
            case SDL_KEYDOWN:
            {
                switch (event.key.keysym.sym)
                {
                case SDLK_a:
                    std::cout << "A pressed\n";
                    camera.transform(glm::translate(glm::mat4(1.0f), glm::vec3(-1.0f, 0.0f, 0.0f)));
                    break;
                case SDLK_d:
                    camera.transform(glm::translate(glm::mat4(1.0f), glm::vec3(1.0f, 0.0f, 0.0f)));
                    break;
                case SDLK_w:
                    camera.transform(glm::translate(glm::mat4(1.0f), glm::vec3(0.0f, 1.0f, 0.0f)));
                    break;
                case SDLK_s:
                    camera.transform(glm::translate(glm::mat4(1.0f), glm::vec3(0.0f, -1.0f, 0.0f)));
                    break;
                default:
                    break;
                }
                isNeedToRedraw = true;
                break;
            }
            }
        }

        if (isNeedToRedraw)
        {
            surf.draw();
            isNeedToRedraw = false;
        }

        frame++;
        if(update.get_ticks() > 1000)
        {
            std::stringstream caption;
            caption << "Average Frames Per Second: " << frame / ( fps.get_ticks() / 1000.f );
            SDL_SetWindowTitle(window, caption.str().c_str());
            update.start();
        }
    }

    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}
